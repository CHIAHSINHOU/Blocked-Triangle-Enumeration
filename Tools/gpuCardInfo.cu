
#include <hip/hip_runtime.h>
#include<cstdio>

int main(){
    int deviceNum;
    hipGetDeviceCount(&deviceNum);
    printf("total %d cards\n", deviceNum);
    for(int i = 0; i < deviceNum; i++){
        struct hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("card %d: %s\n", i, prop.name);
    }

    return 0;
}
