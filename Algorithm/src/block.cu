#include "hip/hip_runtime.h"
#include "block.h"
#include <thrust/device_vector.h>
#include <thrust/sort.h>

void initListArrBlock(
    const EdgeMatrix &edgeBlock, const vector< int > &rowWidth, int blockDim, int blockSize,
    ListArrMatrix &listArrBlock
){
    for(int i = 0; i < blockDim; i++){
        listArrBlock[i] = ListArrRow(blockDim);
    }
    ListArray *d_listArr;
    int *d_nodeArr, *d_edgeArr;
    hipMalloc((void**)&d_listArr, sizeof(ListArray));

    int *offset = new int[blockDim];
    offset[0] = 0;
    for(int i = 1; i < blockDim; i++){
        offset[i] = offset[i-1] + rowWidth[i-1];
        offset[i] *= blockSize;
    }

    for(int i = 0; i < blockDim; i++){
        for(int j = i; j < blockDim; j++){
            thrust::device_vector< Edge > d_edge = edgeBlock[i][j];
            thrust::sort(d_edge.begin(), d_edge.end());

            int nodeNum = rowWidth[i]*blockSize;
            int edgeNum = (int)edgeBlock[i][j].size();
            listArrBlock[i][j].initArray(nodeNum, edgeNum);
            if(listArrBlock[i][j].edgeNum == 0){
                setEmptyArray(nodeNum, listArrBlock[i][j].nodeArr);
                continue;
            }

            Edge *pd_edge = thrust::raw_pointer_cast(d_edge.data());

            int gpuBlock = edgeNum/1024;
            int gpuThread = (edgeNum<1024) ? edgeNum : 1024;
            relabelBlock<<< gpuBlock, gpuThread >>>(edgeNum, offset[i], offset[j], pd_edge);

            hipMalloc((void**)&d_nodeArr, sizeof(int)*(nodeNum+1));
            hipMalloc((void**)&d_edgeArr, sizeof(int)*edgeNum);
            hipMemcpy(&(d_listArr->nodeArr), &d_nodeArr, sizeof(int*), H2D);
            hipMemcpy(&(d_listArr->edgeArr), &d_edgeArr, sizeof(int*), H2D);

            gpuBlock = nodeNum/1024;
            gpuThread = (nodeNum<1024) ? nodeNum : 1024;
            edge2listArr<<< gpuBlock, gpuThread >>>(pd_edge, nodeNum, edgeNum, d_listArr);

            hipMemcpy(listArrBlock[i][j].nodeArr, d_nodeArr, sizeof(int)*(nodeNum+1), D2H);
            hipMemcpy(listArrBlock[i][j].edgeArr, d_edgeArr, sizeof(int)*edgeNum, D2H);

            hipFree(d_nodeArr);
            hipFree(d_edgeArr);
        }
    }

    hipFree(d_listArr);
    delete [] offset;
}

__global__ void relabelBlock(int edgeNum, int uOffset, int vOffset, Edge *edge){
    if(uOffset == 0 && vOffset == 0) return;
    int idx = blockDim.x*blockIdx.x + threadIdx.x;
    int threads = blockDim.x * gridDim.x;

    for(int i = idx; i < edgeNum; i+=threads){
        edge[i].u -= uOffset;
        edge[i].v -= vOffset;
    }
}

