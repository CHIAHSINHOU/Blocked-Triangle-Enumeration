#include "hip/hip_runtime.h"
#include "block.h"
#include <thrust/device_vector.h>
#include <thrust/sort.h>

void initListArrBlock(
    const EdgeMatrix &edgeBlock, const vector< int > &rowWidth, int blockDim, int blockSize,
    ListArrMatrix &listArrBlock
){
    for(int i = 0; i < blockDim; i++){
        listArrBlock[i] = ListArrRow(blockDim);
    }
    ListArray *d_listArr;
    int *d_nodeArr, *d_edgeArr;
    hipMalloc((void**)&d_listArr, sizeof(ListArray));

    int *offset = new int[blockDim];
    offset[0] = 0;
    for(int i = 1; i < blockDim; i++){
        offset[i] = offset[i-1] + rowWidth[i-1];
        offset[i] *= blockSize;
    }

    for(int i = 0; i < blockDim; i++){
        for(int j = i; j < blockDim; j++){
            thrust::device_vector< Edge > d_edge = edgeBlock[i][j];
            thrust::sort(d_edge.begin(), d_edge.end());

            int nodeNum = rowWidth[i]*blockSize;
            int edgeNum = (int)edgeBlock[i][j].size();
            Edge *pd_edge = thrust::raw_pointer_cast(d_edge.data());
            relabelBlock<<< 1, 1 >>>(edgeNum, offset[i], offset[j], pd_edge);

            hipMalloc((void**)&d_nodeArr, sizeof(int)*(nodeNum+1));
            hipMalloc((void**)&d_edgeArr, sizeof(int)*edgeNum);
            hipMemcpy(&(d_listArr->nodeArr), &d_nodeArr, sizeof(int*), H2D);
            hipMemcpy(&(d_listArr->edgeArr), &d_edgeArr, sizeof(int*), H2D);
            edge2listArr<<< 1, 1 >>>(pd_edge, nodeNum, edgeNum, d_listArr);

            listArrBlock[i][j].initArray(nodeNum, edgeNum);
            hipMemcpy(listArrBlock[i][j].nodeArr, d_nodeArr, sizeof(int)*(nodeNum+1), D2H);
            hipMemcpy(listArrBlock[i][j].edgeArr, d_edgeArr, sizeof(int)*edgeNum, D2H);

            hipFree(d_nodeArr);
            hipFree(d_edgeArr);
        }
    }

    hipFree(d_listArr);
    delete [] offset;
}

__global__ void relabelBlock(int edgeNum, int uOffset, int vOffset, Edge *edge){
    if(edgeNum == 0) return;
    if(uOffset == 0 && vOffset == 0) return;
    for(int i = 0; i < edgeNum; i++){
        edge[i].u -= uOffset;
        edge[i].v -= vOffset;
    }
}

