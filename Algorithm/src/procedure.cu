#include<cstdio>
#include <cstdlib>
#include "io.h"
#include "reorder.h"
#include "solve.h"
#include "timer.h"
#include "threadHandler.h"


int assignProc, currTid, threadNum, blockNum;
pthread_t threads[MAX_THREAD_NUM];
bool threadUsed[MAX_THREAD_NUM];
pthread_mutex_t lock;
long long triNum;


int main(int argc, char *argv[]){
    if(argc != 3 && argc != 4 && argc != 6){
        fprintf(stderr, "usage: proc <assign_proc> <input_path> <reorder_or_not> <thread_per_block> <block_num>\n");
        return 0;
    }

    extern int assignProc, threadNum, blockNum;
    assignProc = atoi(argv[1]);
    bool reorder = true;

    if(argc >= 4) reorder = (strcmp("true",argv[3])==0) ? true : false;
    if(assignProc < LIST || assignProc > G_MAT){
        fprintf(stderr, "algo choice\n0: forward\n1: g_forward\n2: mat\n3: g_mat\n");
        return 0;
    }
    if(assignProc == G_LIST || assignProc == G_MAT){
        if(argc != 6){
            fprintf(stderr, "use default %d blocks, %d threads\n", GPU_BLOCK_NUM, GPU_THREAD_NUM);
            blockNum = GPU_BLOCK_NUM;
            threadNum = GPU_THREAD_NUM;
        }
        else{
            blockNum = atoi(argv[5]);
            threadNum = atoi(argv[4]);
        }
    }

    vector< Edge > edge;
    int nodeNum = inputEdge(argv[2], edge);
    
    // resolve first cuda call slow timing issue
    hipFree(0);

    timerInit(1)
    timerStart(0)
    forwardReorder(nodeNum, edge, reorder);

    ListArray listArr, *d_listArr;
    hipMalloc((void**)&d_listArr, sizeof(ListArray));
    if(assignProc == 0)
        cTransBlock(edge, nodeNum, 0, 0, listArr);
    else
        gTransBlock(edge, nodeNum, 0, 0, listArr, d_listArr);
    hipFree(d_listArr);

    pthread_mutex_init(&lock, NULL);

    BitMat::createMask();
    currTid = 0;
    triNum = 0;
    memset(threadUsed, false, MAX_THREAD_NUM);
    scheduler(listArr, listArr, nodeNum, false);

    for(int i = 0; i < MAX_THREAD_NUM; i++) waitThread(i);

    pthread_mutex_destroy(&lock);
    timerEnd("total", 0)

    printf("total triangle: %lld\n", triNum);
    return 0;
}
