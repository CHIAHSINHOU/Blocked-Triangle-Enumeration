#include<cstdio>
#include <cstdlib>
#include "io.h"
#include "reorder.h"
#include "solve.h"
#include "timer.h"
#include "threadHandler.h"


int assignProc, currTid, threadNum, blockNum;
pthread_t threads[MAX_THREAD_NUM];
bool threadUsed[MAX_THREAD_NUM];
pthread_mutex_t lock;
long long triNum;


int main(int argc, char *argv[]){
    if(argc != 3 && argc != 5){
        fprintf(stderr, "usage: proc <assign_proc> <input_path> <thread_per_block> <block_num>\n");
        return 0;
    }

    extern int assignProc, threadNum, blockNum;
    assignProc = atoi(argv[1]);

    if(assignProc < LIST || assignProc > G_MAT){
        fprintf(stderr, "algo choice\n0: forward\n1: g_forward\n2: mat\n3: g_mat\n");
        return 0;
    }
    if(assignProc == G_LIST || assignProc == G_MAT){
        if(argc != 5){
            fprintf(stderr, "use default %d blocks, %d threads\n", GPU_BLOCK_NUM, GPU_THREAD_NUM);
            blockNum = GPU_BLOCK_NUM;
            threadNum = GPU_THREAD_NUM;
        }
        else{
            blockNum = atoi(argv[4]);
            threadNum = atoi(argv[3]);
        }
    }

    vector< Edge > edge;
    int nodeNum = inputEdge(argv[2], edge);
    
    // resolve first cuda call slow timing issue
    hipFree(0);

    timerInit(1)
    timerStart(0)
    int edgeNum = (int)edge.size();
    double density = (double)edgeNum/((double)nodeNum*nodeNum/2.0) * 100.0;

    if(density > 0.01)
        cForwardReorder(nodeNum, edge);

    ListArray listArr, *d_listArr;
    hipMalloc((void**)&d_listArr, sizeof(ListArray));
    transBlock(edge, nodeNum, 0, 0, listArr, d_listArr);

    pthread_mutex_init(&lock, NULL);

    BitMat::createMask();
    currTid = 0;
    triNum = 0;
    memset(threadUsed, false, MAX_THREAD_NUM);
    scheduler(listArr, listArr, nodeNum, false);

    for(int i = 0; i < MAX_THREAD_NUM; i++) waitThread(i);

    pthread_mutex_destroy(&lock);

    timerEnd("time", 0)

//    printf("%d node, %d edge, density = %lf%%\n", nodeNum, edgeNum, density);
    printf("total triangle: %lld\n", triNum);
    return 0;
}
