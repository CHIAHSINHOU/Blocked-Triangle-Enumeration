#include "hip/hip_runtime.h"
#include "list.h"
#include "binaryTree.h"

long long gpuCountTriangle(const ListArg &listArg){
    const ListArray &edge = listArg.edge;
    const ListArray &target = listArg.target;
    int maxDeg = listArg.maxDeg;
    int threadNum = listArg.threadNum;
    int blockNum = listArg.blockNum;

    long long *d_triNum, triNum;
    ListArray *d_edge, *d_target;
    int *d_edge_edgeArr, *d_edge_nodeArr, *d_target_edgeArr, *d_target_nodeArr;
    
    hipMalloc((void**)&d_triNum, sizeof(long long)*blockNum);

    // copy edge to device
    hipMalloc((void**)&d_edge, sizeof(ListArray));
    hipMemcpy(d_edge, &edge, sizeof(ListArray), H2D);
    // edge.nodeArr
    hipMalloc((void**)&d_edge_nodeArr, sizeof(int)*(edge.nodeNum+1));
    hipMemcpy(d_edge_nodeArr, edge.nodeArr, sizeof(int)*(edge.nodeNum+1), H2D);
    hipMemcpy(&(d_edge->nodeArr), &d_edge_nodeArr, sizeof(int*), H2D);
    // edge.edgeArr
    hipMalloc((void**)&d_edge_edgeArr, sizeof(int)*edge.edgeNum);
    hipMemcpy(d_edge_edgeArr, edge.edgeArr, sizeof(int)*edge.edgeNum, H2D);
    hipMemcpy(&(d_edge->edgeArr), &d_edge_edgeArr, sizeof(int*), H2D);

    // copy target to device
    hipMalloc((void**)&d_target, sizeof(ListArray));
    hipMemcpy(d_target, &target, sizeof(ListArray), H2D);
    // target.nodeArr
    hipMalloc((void**)&d_target_nodeArr, sizeof(int)*(target.nodeNum+1));
    hipMemcpy(d_target_nodeArr, target.nodeArr, sizeof(int)*(target.nodeNum+1), H2D);
    hipMemcpy(&(d_target->nodeArr), &d_target_nodeArr, sizeof(int*), H2D);
    // target.edgeArr
    hipMalloc((void**)&d_target_edgeArr, sizeof(int)*target.edgeNum);
    hipMemcpy(d_target_edgeArr, target.edgeArr, sizeof(int)*target.edgeNum, H2D);
    hipMemcpy(&(d_target->edgeArr), &d_target_edgeArr, sizeof(int*), H2D);

    int smSize = maxDeg*sizeof(int);
    gpuCountList<<< blockNum, threadNum, smSize >>>(d_edge, d_target, d_triNum);
    sumTriangle<<< 1, 1 >>>(d_triNum, blockNum);
    hipMemcpy(&triNum, d_triNum, sizeof(long long), D2H);

    hipFree(d_triNum);
    hipFree(d_edge);
    hipFree(d_edge_edgeArr);
    hipFree(d_edge_nodeArr);
    hipFree(d_target);
    hipFree(d_target_edgeArr);
    hipFree(d_target_nodeArr);

    return triNum;
}

__global__ void gpuCountList(const ListArray *edge, const ListArray *target, long long *triNum){
    __shared__ long long threadTriNum[1024];
    int bound = nearestLessPowOf2(blockDim.x);

    triNum[blockIdx.x] = 0;
    // iterator through each edge (u, v)
    int range = edge->nodeNum;
    for(int u = blockIdx.x; u < range; u += gridDim.x){
        int uLen = target->getDeg(u);
        const int *uList = target->neiStart(u);
        
        // move node u's adj list (in target) to shared memory
        int uDeg = edge->getDeg(u);
        for(int i = threadIdx.x; i < uLen; i += blockDim.x){
            uAdj[i] = uList[i];
        }
        __syncthreads();

        // counting triangle number
        threadTriNum[threadIdx.x] = 0;
        const int *uNei = edge->neiStart(u);
        for(int i = threadIdx.x; i < uDeg; i += blockDim.x){
            int v = uNei[i];
            int vLen = target->getDeg(v);
            const int *vList = target->neiStart(v);
            // intersect u list and v list in target
            threadTriNum[threadIdx.x] += intersectList(uLen, vLen, uAdj, vList);
        }
        __syncthreads();

        binaryTreeSum(threadTriNum, blockDim.x, bound);
        if(threadIdx.x==0)
            triNum[blockIdx.x] += threadTriNum[0];

//        if(threadIdx.x==0)
//            triNum[blockIdx.x] += linearSum(threadTriNum, blockDim.x);

        __syncthreads();
    }
}

__host__ __device__ long long intersectList(int sz1, int sz2, const int *l1, const int *l2){
    long long triNum = 0;
    for(int i = sz1-1, j = sz2-1; i >= 0 && j >= 0;){
        if(l1[i] > l2[j]) i--;
        else if(l1[i] < l2[j]) j--;
        else{
            i--, j--;
            triNum++;
        }
    }
    return triNum;
}

