#include<cstdio>
#include<cstdlib>
#include "io.h"
#include "reorder.h"
#include "solve.h"
#include "block.h"
#include "timer.h"
#include "mat.h"
#include "degeneracy.h"

int assignProc, blockNum, threadNum;
double densityBoundary;
int edgeNumLimit;
long long triNum;
UI mask[BIT_PER_ENTRY], *d_mask;
UC oneBitNum[BIT_NUM_TABLE_SIZE], *d_oneBitNum;

int main(int argc, char *argv[]){
    if(argc < 2){
        fprintf(stderr, "arguments:\n");
        fprintf(stderr, "  input_path\t\t\t(required)\n");
        fprintf(stderr, "  density_boundary\t\t(default=0.06)\n");
        fprintf(stderr, "  block_size\t\t\t(default=3072)\n");
        fprintf(stderr, "  edge_num_limit\t\t(default=20M)\n");
        fprintf(stderr, "  reorder_or_not\t\t(default=true)\n");
        fprintf(stderr, "  assign_proc\t\t\t(default:auto)\n");
        fprintf(stderr, "  max_allowed_gpu_block_num\t(default=16000)\n");
        fprintf(stderr, "  max_allowed_thread_per_block\t(default=1024\n");
        return 0;
    }

    densityBoundary = DENSITY_BOUNDARY;
    int blockSize = DEFAULT_BLOCK_SIZE;
    edgeNumLimit = EDGE_NUM_LIMIT;
    bool reorder = true;
    assignProc = UNDEF;
    blockNum = GPU_BLOCK_NUM;
    threadNum = GPU_THREAD_NUM;
    if(argc >= 3) densityBoundary = atof(argv[2]);
    if(argc >= 4) blockSize = atoi(argv[3]);
    if(argc >= 5) edgeNumLimit = atoi(argv[4]);
    if(argc >= 6) reorder = (strcmp("true",argv[5])==0) ? true : false;
    if(argc >= 7) assignProc = atoi(argv[6]);
    if(argc >= 8) blockNum = atoi(argv[7]);
    if(argc >= 9) threadNum = atoi(argv[8]);

    vector< Edge > edge;

    int nodeNum = inputEdge(argv[1], edge);

    // resolve first cuda call slow timing issue
    hipFree(0);

    timerInit(2)
    timerStart(0)

    int edgeNum = (int)edge.size();
    double density = (double)edgeNum/((double)nodeNum*nodeNum/2.0) * 100.0;

    timerStart(1)
    //forwardReorder(nodeNum, edge, reorder);
    reorderByDegeneracy(nodeNum, edge);
    timerEnd("reorder", 1)

    timerStart(1)
    createMask(mask, &d_mask);
    createOneBitNumTable(oneBitNum, &d_oneBitNum);
    triNum = 0;
    timerEnd("initial", 1)

/*    if(edgeNum <= EDGE_NUM_LIMIT){
        timerStart(1)
        ListArray listArr, *d_listArr;
        hipMalloc((void**)&d_listArr, sizeof(ListArray));
        gTransBlock(edge, nodeNum, 0, 0, listArr, d_listArr);
        hipFree(d_listArr);
        timerEnd("edge->list", 1)

        timerStart(1)
        scheduler(listArr, listArr, nodeNum, true);
        timerEnd("count", 1)
    }
    else{*/
        EdgeMatrix edgeBlock;
        vector< int > rowWidth;
        int remain = nodeNum % blockSize;
        if(remain == 0) remain = blockSize;
        int blockDim = initEdgeBlock(edge, nodeNum, blockSize, remain, edgeBlock, rowWidth);
        rowWidth.resize(blockDim);
        printf("divide into %d subgraph(s):", blockDim);
        for(int i = 0; i < (int)rowWidth.size(); i++){
            printf(" %d", rowWidth[i]);
        }

        timerStart(1)
        ListArrMatrix listArrBlock(blockDim);
        initListArrBlock(edgeBlock, rowWidth, blockDim, blockSize, listArrBlock);
        timerEnd("edge->list", 1)

        timerStart(1)
        findTriangle(listArrBlock, rowWidth, blockDim);
        timerEnd("count", 1)
//    }

    timerEnd("total", 0)
    hipFree(d_oneBitNum);
    hipFree(d_mask);

    fprintf(stderr, "%d node, %d edge, density = %lf%%\n", nodeNum, edgeNum, density);
    printf("total triangle: %lld\n", triNum);
    return 0;
}
