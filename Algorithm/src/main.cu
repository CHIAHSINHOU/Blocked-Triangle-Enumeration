#include<cstdio>
#include<cstdlib>
#include "io.h"
#include "reorder.h"
#include "solve.h"
#include "block.h"
#include "threadHandler.h"
#include "timer.h"
#include "mat.h"

int assignProc, currTid;
int blockNum, threadNum;
pthread_t threads[MAX_THREAD_NUM];
bool threadUsed[MAX_THREAD_NUM];
pthread_mutex_t lock;
long long triNum;
unsigned char oneBitNum[BIT_NUM_TABLE_SIZE];

int main(int argc, char *argv[]){
    if(argc < 3){
        fprintf(stderr, "usage: count <input_path> <block_size> (<reorder_or_not> <assign_proc> <block_num> <thread_num>)\n");
        return 0;
    }

    int blockSize = atoi(argv[2]);
    bool reorder = true;
    assignProc = UNDEF;
    blockNum = GPU_BLOCK_NUM;
    threadNum = GPU_THREAD_NUM;
    if(argc >= 4) reorder = (strcmp("true",argv[3])==0) ? true : false;
    if(argc >= 5) assignProc = atoi(argv[4]);
    if(argc >= 6) blockNum = atoi(argv[5]);
    if(argc >= 7) threadNum = atoi(argv[6]);

    vector< Edge > edge;

    int nodeNum = inputEdge(argv[1], edge);

    // resolve first cuda call slow timing issue
    hipFree(0);

    timerInit(1)
    timerStart(0)

    int edgeNum = (int)edge.size();
//    double density = (double)edgeNum/((double)nodeNum*nodeNum/2.0) * 100.0;

    //timerStart(1)
    forwardReorder(nodeNum, edge, reorder);
    //timerEnd("reorder", 1)

    BitMat::createMask();
    createOneBitNumTable(oneBitNum);
    currTid = 0;
    triNum = 0;
    memset(threadUsed, false, MAX_THREAD_NUM);

    pthread_mutex_init(&lock, NULL);

    if(edgeNum <= EDGE_NUM_LIMIT){
        ListArray listArr, *d_listArr;
        hipMalloc((void**)&d_listArr, sizeof(ListArray));
        gTransBlock(edge, nodeNum, 0, 0, listArr, d_listArr);
        hipFree(d_listArr);

        //timerStart(1)
        scheduler(listArr, listArr, nodeNum, false);
        for(int i = 0; i < MAX_THREAD_NUM; i++) waitThread(i);
        //timerEnd("count", 1)
    }
    else{
        EdgeMatrix edgeBlock;
        vector< int > rowWidth;
        int blockDim = initEdgeBlock(edge, nodeNum, blockSize, edgeBlock, rowWidth);
        rowWidth.resize(blockDim);
//        fprintf(stderr, "blockDim: %d\n", blockDim);
        for(int i = 0; i < (int)rowWidth.size(); i++){
            rowWidth[i] *= blockSize;
//            printf("%d %d\n", i, rowWidth[i]);
        }

        ListArrMatrix listArrBlock(blockDim);
        initListArrBlock(edgeBlock, rowWidth, blockDim, blockSize, listArrBlock);

        //timerStart(1)
        findTriangle(listArrBlock, rowWidth, blockDim);
        //timerEnd("count", 1)
    }

    pthread_mutex_destroy(&lock);
    timerEnd("total", 0)

//    fprintf(stderr, "%d node, %d edge, density = %lf%%\n", nodeNum, edgeNum, density);
    printf("total triangle: %lld\n", triNum);
    return 0;
}
