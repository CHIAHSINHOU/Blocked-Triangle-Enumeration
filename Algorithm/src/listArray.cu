#include "hip/hip_runtime.h"
#include "listArray.h"
#include<cstring>
#include<cstdio>

ListArray::~ListArray(void){
    delete [] nodeArr;
    delete [] edgeArr;
}

void ListArray::initArray(int n, int e){
    nodeNum = n, edgeNum = e;
    nodeArr = new int[sizeof(int)*(n+1)];
    edgeArr = new int[sizeof(int)*e];
}

int ListArray::getMaxDegree(void) const{
    int mmax = 0;
    for(int i = 0; i < nodeNum; i++){
        int deg = nodeArr[i+1] - nodeArr[i];
        if(deg > mmax) mmax = deg;
    }
    return mmax;
}

DECORATE  int ListArray::getDeg(int v) const{
    if(v < 0 || v >= nodeNum) return 0;
    return nodeArr[v+1] - nodeArr[v];
}

DECORATE int ListArray::getNodeNum() const{
    return nodeNum;
}

DECORATE const int* ListArray::neiStart(int v) const{
    if(v < 0 || v >= nodeNum) return NULL;
    if(nodeArr[v] >= edgeNum) return NULL;
    return &edgeArr[nodeArr[v]];
}

void ListArray::integrate(const ListArray &a, ListArray &res) const{
    res.initArray(nodeNum+a.nodeNum, edgeNum+a.edgeNum);
    for(int i = 0; i < nodeNum; i++)
        res.nodeArr[i] = nodeArr[i];
    for(int i = 0, j = nodeNum; i < a.nodeNum; i++, j++)
        res.nodeArr[j] = a.nodeArr[i] + edgeNum;
    res.nodeArr[res.nodeNum] = res.edgeNum;

    for(int i = 0; i < edgeNum; i++)
        res.edgeArr[i] = edgeArr[i];
    for(int i = 0, j = edgeNum; i < a.edgeNum; i++, j++)
        res.edgeArr[j] = a.edgeArr[i];
}

DECORATE void ListArray::print(void) const{
    printf("node:");
    for(int i = 0; i <= nodeNum; i++){
        printf(" %d", nodeArr[i]);
    }
    printf("\nedge:");
    for(int i = 0; i < edgeNum; i++){
        printf(" %d", edgeArr[i]);
    }
    printf("\n");
}

__global__ void initNodeArr(int nodeNum, ListArray *listArr){
    int idx = blockDim.x*blockIdx.x + threadIdx.x;
    int threads = blockDim.x * gridDim.x;

    for(int i = idx; i < nodeNum; i+=threads)
        listArr->nodeArr[i] = -1;
}

__global__ void edge2listArr(const Edge *edge, int nodeNum, int edgeNum, ListArray *listArr){
    int idx = blockDim.x*blockIdx.x + threadIdx.x;
    int threads = blockDim.x * gridDim.x;

    for(int i = idx; i < edgeNum; i+=threads){
        listArr->edgeArr[i] = edge[i].v;
        if(i > 0){
            if(edge[i-1].u != edge[i].u)
                listArr->nodeArr[edge[i].u] = i;
        }
    }
    listArr->nodeArr[edge[0].u] = 0;
    listArr->nodeArr[nodeNum] = edgeNum;
}

__global__ void removeEmptyFlag(int nodeNum, ListArray *listArr){
    int idx = blockDim.x*blockIdx.x + threadIdx.x;
    int threads = blockDim.x * gridDim.x;

    for(int i = idx; i <= nodeNum; i+=threads){
        if(listArr->nodeArr[i] != -1 && i > 0){
            for(int j = i-1; j >= 0 && listArr->nodeArr[j]==-1; j--){
                listArr->nodeArr[j] = listArr->nodeArr[i];
            }
        }
    }
}
