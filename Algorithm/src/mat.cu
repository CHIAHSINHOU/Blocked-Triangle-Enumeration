#include "hip/hip_runtime.h"
#include "mat.h"
#include "binaryTree.h"

#include<cstdio>

long long gpuCountTriangleMat(UI *mat, int entryNum, int nodeNum, int threadNum, int blockNum){
    long long *d_triNum, triNum;
    UI *d_mat;

    hipMalloc((void**)&d_triNum, sizeof(long long)*blockNum);
    hipMalloc((void**)&d_mat, sizeof(UI)*entryNum*nodeNum);
    hipMemcpy(d_mat, mat, sizeof(UI)*entryNum*nodeNum, hipMemcpyHostToDevice);

    int smSize = nodeNum*sizeof(UI);
    gpuCountMat<<< blockNum, threadNum, smSize >>>(d_mat, entryNum, nodeNum, d_triNum, threadNum, blockNum);
    hipDeviceSynchronize();

    sumTriangle<<< 1, 1 >>>(d_triNum, blockNum);
    hipDeviceSynchronize();
    hipMemcpy(&triNum, d_triNum, sizeof(long long), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    hipFree(d_triNum);
    hipFree(d_mat);

    return triNum/6;
}

__global__ void gpuCountMat(UI *mat, int entryNum, int nodeNum, long long *triNum, int threadNum, int blockNum){
    __shared__ long long threadTriNum[1024];
    int bound = nearestLessPowOf2(blockDim.x);

    triNum[blockIdx.x] = 0;

    for(int e = blockIdx.x; e < entryNum; e += gridDim.x){

        // move tile area to shared memory
        for(int i = threadIdx.x; i < nodeNum; i++){
            tile[i] = mat[i*entryNum+e];
        }
        __syncthreads();

        // count triangle number
        threadTriNum[threadIdx.x] = 0;
        for(int i = threadIdx.x; i < nodeNum; i += blockDim.x){
            // iterator through each entry of the row
            for(int j = 0; j < entryNum; j++){
                // iterator through each bit
                UI content = mat[i*entryNum+j];
                for(int k = j*BIT_PER_ENTRY; content > 0; k++, content/=2){
                    if(content % 2 == 1){ // edge(i, k) exists
//                        threadTriNum[threadIdx.x] += andList(mat, i, k, e, entryNum);
                        threadTriNum[threadIdx.x] += andList(tile, i, k, 0, 1);
                    }
                }
            }
        }
        __syncthreads();

        binaryTreeSum(threadTriNum, blockDim.x, bound);
        if(threadIdx.x==0){
            triNum[blockIdx.x] += threadTriNum[0];
        }

/*        if(threadIdx.x==0)
            triNum[blockIdx.x] += linearSum(threadTriNum, blockDim.x);*/

        __syncthreads();
    }
}

__host__ __device__ long long andList(UI *mat, int l1, int l2, int entry, int width){
    long long triNum = 0;
    UI result = mat[l1*width+entry] & mat[l2*width+entry];
    triNum = countOneBits(result);
    return triNum;
}

__host__ __device__ long long countOneBits(UI tar){
    long long ones = 0;
    for(; tar; tar/=2)
        ones += tar % 2;
    return ones;
}

