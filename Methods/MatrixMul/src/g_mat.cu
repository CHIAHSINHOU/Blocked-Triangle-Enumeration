#include<cstdio>
#include<cstdlib>
#include "tools.h"
#include "timer.h"
#include "matFunc.h"


int main(int argc, char *argv[]){
    if(argc != 6){
        fprintf(stderr, "usage: tiledBit <input_path> <node_num> <node_per_tile> <thread_per_block> <block_num>\n");
        return 0;
    }

    timerInit(2)
    timerStart(0)

    int nodeNum = atoi(argv[2]);
    int nodePerTile = atoi(argv[3]);
	int threadPerBlock = atoi(argv[4]);
    int blockNum = atoi(argv[5]);
    if(nodePerTile % BIT_PER_ENTRY != 0){
        fprintf(stderr, "node per tile must be multiple of %lu\n", BIT_PER_ENTRY);
        return 0;
    }
    int entryPerTile = nodePerTile / BIT_PER_ENTRY;

    int entryNum = averageCeil(nodeNum, BIT_PER_ENTRY);
    UI *mat = (UI*)malloc(entryNum*nodeNum*sizeof(UI));

    timerStart(1)
    inputMat(argv[1], mat, entryNum*nodeNum*sizeof(UI), entryNum);
    timerEnd("input", 1)

    int triNum, *d_triNum;
    UI *d_mat;

    timerStart(1)
    initDeviceTriNum((void**)&d_triNum);
    matCopyToDevice(nodeNum, mat, (void**)&d_mat);
    timerEnd("cuda copy", 1)

    timerStart(1)
    int smSize = (entryPerTile*nodeNum + threadPerBlock) * sizeof(UI);
    gpuCountTriNum<<< blockNum, threadPerBlock, smSize >>>(d_mat, d_triNum, nodeNum, nodePerTile);
    hipDeviceSynchronize();
    timerEnd("find triangle", 1)

    hipMemcpy(&triNum, d_triNum, sizeof(int), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
    printf("total triangle: %d\n", triNum/6);

    hipFree(d_triNum);
    hipFree(d_mat);
    free(mat);

    timerEnd("total", 0)

    return 0;
}

