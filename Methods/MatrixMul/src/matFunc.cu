#include "hip/hip_runtime.h"
#include "matFunc.h"
#include "tools.h"
#include<cstdio>

void inputMat(const char *inFile, unsigned int *mat, int edgeSize, int entryNum){
    FILE *fp = fopen(inFile, "r");
    int u, v;
    memset(mat, 0, edgeSize);
    while(fscanf(fp, "%d%d", &u, &v) != EOF){
        setEdge(u, v);
        setEdge(v, u);
    }
    fclose(fp);
}

void matCopyToDevice(int nodeNum, void* mat, void* d_mat){
    int entryNum = averageCeil(nodeNum, BIT_PER_ENTRY);
    hipMemcpy(d_mat, mat, entryNum*nodeNum*sizeof(UI), hipMemcpyHostToDevice);
}

int cpuCountTriNum(int nodeNum, int nodePerTile, UI *mat){
    int triNum = 0;
    int round = averageCeil(nodeNum, nodePerTile);
    int entryNum = averageCeil(nodeNum, BIT_PER_ENTRY);
    int entryPerTile = nodePerTile / BIT_PER_ENTRY;
    for(int t = 0; t < round; t++){
        int st = t * entryPerTile;
        int ed = st + entryPerTile;
        if(ed > entryNum) ed = entryNum;
        for(int i = 0; i < nodeNum; i++){
            for(int j = i+1; j < nodeNum; j++){
                if(!getEdge(i, j)) continue;
                for(int k = st; k < ed; k++){
                    UI result = mat[i*entryNum+k] & mat[j*entryNum+k];
                    triNum += countOneBits(result);
                }
            }
        }
    }
    return triNum/3;
}

__global__ void gpuCountTriNum(UI *mat, int *triNum, int nodeNum, int nodePerTile){
    int entryNum = averageCeil(nodeNum, BIT_PER_ENTRY);
    int entryPerTile = nodePerTile / BIT_PER_ENTRY;
    int nodePerThread = averageCeil(nodeNum, blockDim.x);

    int tileNum = averageCeil(nodeNum, nodePerTile);
    int tilePerBlock = averageCeil(tileNum, gridDim.x);
    for(int r = 0; r < tilePerBlock; r++){
        int tileID = blockIdx.x*tilePerBlock + r;
        if(tileID >= tileNum) continue;
        int offset = tileID * entryPerTile;
        int tileLen = entryPerTile;
        if(offset+tileLen > entryNum) tileLen = entryNum - offset;

        // move adj matrix tiled area to shared memory
        for(int i = 0; i < nodePerThread; i++){
            int idx = threadIdx.x*nodePerThread + i;
            if(idx >= nodeNum) continue;
            for(int j = 0; j < tileLen; j++){
                shared[idx*entryPerTile+j] = mat[idx*entryNum+j+offset]; // adjMat[idx][j]
            }
        }
        __syncthreads();

        // counting triangle number
        int tileSize = entryPerTile*nodeNum;
        int tid = tileSize + threadIdx.x;
        shared[tid] = 0; // threadTriNum[tid]
        for(int i = 0; i < nodePerThread; i++){
            int idx = threadIdx.x*nodePerThread + i;
            if(idx >= nodeNum) continue;
            for(int j = 0; j < nodeNum; j++){
                if(idx == j || !getEdge(idx, j)) continue;
                for(int k = 0; k < tileLen; k++){
                    UI result = shared[idx*entryPerTile+k] & shared[j*entryPerTile+k];
                    shared[tid] += countOneBits(result); //threadTriNum[tid]
                }
            }
        }
        __syncthreads();

        if(threadIdx.x == 0){
            sumTriangle(triNum, (int*)&shared[tileSize]);
        }
    }
}

__host__ __device__ int countOneBits(UI tar){
    int ones = 0;
    for(; tar; tar/=2)
        ones += tar % 2;
    return ones;
}

