
#include <hip/hip_runtime.h>
#include<cstdio>
#include<cstdlib>
#include<cstring>
#include<cmath>
#include<sys/time.h>

#define BIT_PER_ENTRY (sizeof(UI)*8)

#define getEdge(u,v) (((UI)1<<(v%BIT_PER_ENTRY))&edge[u*entryNum+(v/BIT_PER_ENTRY)])>0
#define setEdge(u,v) edge[u*entryNum+(v/BIT_PER_ENTRY)]|=((UI)1<<(v%BIT_PER_ENTRY))

#define cntTime(st,ed)\
((double)ed.tv_sec*1000000+ed.tv_usec-(st.tv_sec*1000000+st.tv_usec))/1000

#define timerInit(n)\
struct timeval st[n], ed[n];

#define timerStart(n)\
gettimeofday(&st[n], NULL);

#define timerEnd(tar, n)\
gettimeofday(&ed[n], NULL);\
//fprintf(stderr, " %.3lf", cntTime(st[n],ed[n]));
//fprintf(stderr, "%s: %.3lf ms\n", tar, cntTime(st[n],ed[n]));

typedef unsigned int UI;

__global__ void countTriNum(UI *edge, int *triNum, int nodeNum, int nodePerTile);
__device__ int countOneBits(UI tar);

int main(int argc, char *argv[]){
    if(argc != 5){
        fprintf(stderr, "usage: tiledBit <input_path> <node_num> <node_per_tile> <thread_per_block>\n");
        return 0;
    }

    timerInit(2)
    timerStart(0)

    int nodeNum = atoi(argv[2]);
    int nodePerTile = atoi(argv[3]);
	int threadPerBlock = atoi(argv[4]);
    if(nodePerTile % BIT_PER_ENTRY != 0){
        fprintf(stderr, "node per tile must be multiple of %lu\n", BIT_PER_ENTRY);
        return 0;
    }

    int entryNum = (int)ceil((double)nodeNum/BIT_PER_ENTRY-0.001);
	int tileNum = (int)ceil((double)nodeNum/nodePerTile-0.001);
    UI *edge = (UI*)malloc(entryNum*nodeNum*sizeof(UI));

    timerStart(1)
    FILE *fp = fopen(argv[1], "r");
    int u, v;
    memset(edge, 0, entryNum*nodeNum*sizeof(UI));
    while(fscanf(fp, "%d%d", &u, &v) != EOF){
        setEdge(u, v);
        setEdge(v, u);
    }
    fclose(fp);
    timerEnd("input", 1)

    int triNum = 0, *d_triNum;
    UI *d_edge;

    timerStart(1)
    hipMalloc((void**)&d_edge, entryNum*nodeNum*BIT_PER_ENTRY);
    hipMalloc((void**)&d_triNum, sizeof(int));
    hipMemcpy(d_edge, edge, entryNum*nodeNum*BIT_PER_ENTRY, hipMemcpyHostToDevice);
    hipMemcpy(d_triNum, &triNum, sizeof(int), hipMemcpyHostToDevice);
    timerEnd("cuda copy", 1)

    timerStart(1)
//    int smSize = entryPerTile * nodeNum * sizeof(UI);
    countTriNum<<< tileNum, threadPerBlock >>>(d_edge, d_triNum, nodeNum, nodePerTile);
    hipDeviceSynchronize();
    timerEnd("find triangle", 1)

    hipMemcpy(&triNum, d_triNum, sizeof(int), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
    printf("total triangle: %d\n", triNum/6);

    hipFree(d_triNum);
    hipFree(d_edge);

    free(edge);

    timerEnd("total", 0)

    return 0;
}

__global__ void countTriNum(UI *edge, int *triNum, int nodeNum, int nodePerTile){
    int entryNum = (int)ceil((double)nodeNum/BIT_PER_ENTRY-0.001);
    int entryPerTile = nodePerTile / BIT_PER_ENTRY;
	int nodePerThread = (int)ceil((double)nodeNum/blockDim.x-0.001);
    int st = blockIdx.x * entryPerTile;
    int ed = st + entryPerTile;
	if(ed > nodeNum) ed = nodeNum;
    for(int i = 0; i < nodePerThread; i++){
        int idx = threadIdx.x*nodePerThread + i;
        if(idx < nodeNum){
	        for(int j = 0; j < nodeNum; j++){
                if(idx == j || !getEdge(idx, j)) continue;
           	    for(int k = st; k < ed; k++){
               	    UI result = edge[idx*entryNum+k] & edge[j*entryNum+k];
                   	atomicAdd(triNum, countOneBits(result));
                }
            }
        }
    }
}

__device__ int countOneBits(UI tar){
    int ones = 0;
    for(; tar; tar/=2)
        ones += tar % 2;
    return ones;
}
